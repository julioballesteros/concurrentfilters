#include "hip/hip_runtime.h"
#pragma once
#include "kernel.h"

using namespace std::chrono;

//La macro es una sola linea.
//Macro for checking cuda errors following a cuda launch or api call
#define cudaCheckError() {                                          \
 hipError_t e=hipGetLastError();                                 \
 if(e!=hipSuccess) {                                              \
   printf("Cuda failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(e));           \
 \
 }                                                                 \
}

__global__ void filter2D_Kernel_1(char** in, char** out, float* mask, int width, int height)
{
	//Seleccionar un pixel de image_in
	int threadIDx = threadIdx.x + blockIdx.x * blockDim.x;
	int threadIDy = threadIdx.y + blockIdx.y * blockDim.y;

	if (threadIDx >= width) return;
	if (threadIDy >= height) return;

	if (threadIDx == 0 || threadIDx == width - 1)	return;
	if (threadIDy == 0 || threadIDy >= height - 1)	return;

	//Aplicar filtro a pixel
	pixel32bpp pixelIn, pixelOut;
	float a, r, g, b;
	a = r = g = b = 0;
	for (int i = -1; i <= 1; i++)
	{
		for (int j = -1; j <= 1; j++)
		{
			pixelIn = *((pixel32bpp*)(&(in[threadIDy + i][(threadIDx + j) * 4])));
			a += ((float)pixelIn.a)*mask[(i + 1) * 3 + j + 1];
			r += ((float)pixelIn.r)*mask[(i + 1) * 3 + j + 1];
			g += ((float)pixelIn.g)*mask[(i + 1) * 3 + j + 1];
			b += ((float)pixelIn.b)*mask[(i + 1) * 3 + j + 1];
		}
	}

	//Guardar resultado en image_out
	pixelOut.r = (char)r;
	pixelOut.g = (char)g;
	pixelOut.b = (char)b;
	pixelOut.a = (char)a;
	((pixel32bpp*)out[threadIDy])[threadIDx] = pixelOut;
}


Image_t* filter2D_CUDA_1(Image_t* img, float* mask, int numThreadsKernel)
{
	//Reservar imagen resultado
	Image_t* img_result = (Image_t*)malloc(sizeof(Image_t));
	img_result->height = img->height;
	img_result->width = img->width;
	img_result->color_type = img->color_type;
	img_result->bit_depth = img->bit_depth;
	img_result->row_pointers = (char**)malloc(sizeof(char*)*img->height);
	for (int i = 0; i < img->height; i++)
	{
		img_result->row_pointers[i] = (char*)malloc(sizeof(char)*img->width * 4);
		memset(img_result->row_pointers[i], 0, sizeof(char)*img->width * 4);
	}

	//Copiar datos a GPU
	high_resolution_clock::time_point t1 = high_resolution_clock::now();
	
		//reservar memoria para la imagen resultado
	char** row_pointers_result = (char**)malloc(sizeof(char*)*img->height);
	for (int i = 0; i < img->height; i++)
	{
		hipMalloc((void**)&row_pointers_result[i], sizeof(char)*img->width * 4);
		hipMemset(row_pointers_result[i], 0, sizeof(char)*img->width * 4);
		cudaCheckError();
	}
	char** d_pixels_result = NULL;
	hipMalloc((void**)&d_pixels_result, sizeof(char*)*img->height);
	hipMemset(d_pixels_result, 0, sizeof(char*)*img->height);
	hipMemcpy(d_pixels_result, row_pointers_result, sizeof(char*)*img->height, hipMemcpyHostToDevice);
	cudaCheckError();

		//reservar memoria y copiar imagen original
	char** row_pointers = (char**)malloc(sizeof(char*)*img->height);
	for (int i = 0; i < img->height; i++)
	{
		hipMalloc((void**)&row_pointers[i], sizeof(char)*img->width * 4);
		hipMemset(row_pointers[i], 0, sizeof(char)*img->width * 4);
		hipMemcpy(row_pointers[i], img->row_pointers[i], sizeof(char)*img->width * 4, hipMemcpyHostToDevice);
		cudaCheckError();
	}
	char** d_pixels = NULL;
	hipMalloc((void**)&d_pixels, sizeof(char*)*img->height);
	hipMemset(d_pixels, 0, sizeof(char*)*img->height);
	hipMemcpy(d_pixels, row_pointers, sizeof(char*)*img->height, hipMemcpyHostToDevice);
	cudaCheckError();

		//copiar el filtro a GPU
	float* d_mask;
	hipMalloc((void**)&d_mask, 9 * sizeof(float));
	hipMemset(d_mask, 0, sizeof(float)*9);
	hipMemcpy(d_mask, mask, sizeof(float)*9, hipMemcpyHostToDevice);
	cudaCheckError();

	duration<double> time_span = duration_cast<duration<double>>(high_resolution_clock::now() - t1);
	std::cout << "\t Tiempo de copia de datos de CPU a GPU: " << time_span.count() * 1000 << "\n";

	//Ejecutar filter2D_Kernel_1
	dim3 threadsPerBlock = dim3(numThreadsKernel, numThreadsKernel, 1);
	dim3 numBlocks = dim3((img->width / threadsPerBlock.x) + 1, (img->height / threadsPerBlock.y) + 1, 1);

	t1 = high_resolution_clock::now();
	filter2D_Kernel_1 << <numBlocks, threadsPerBlock>> >
		(d_pixels,d_pixels_result, d_mask, img->width, img->height);
	cudaCheckError();
	hipDeviceSynchronize();
	cudaCheckError();
	time_span = duration_cast<duration<double>>(high_resolution_clock::now() - t1);
	std::cout << "\t Tiempo de ejecuci�n del kernel: " << time_span.count() * 1000 << "\n";

	//Copiar datos de GPU
	t1 = high_resolution_clock::now();
	for (int i = 0; i < img->height; i++)
	{
		hipMemcpy(img_result->row_pointers[i], row_pointers_result[i], sizeof(char) * img->width * 4, hipMemcpyDeviceToHost);
	}
	cudaCheckError();
	time_span = duration_cast<duration<double>>(high_resolution_clock::now() - t1);
	std::cout << "\t Tiempo de copia de datos de GPU a CPU: " << time_span.count() * 1000 << "\n";

	//Liberar memoria
	for (int i = 0; i < img->height; i++)
	{
		hipFree(row_pointers_result[i]);
		hipFree(row_pointers[i]);
	}
	free(row_pointers_result);
	free(row_pointers);
	hipFree(d_pixels_result);
	hipFree(d_pixels);
	hipFree(d_mask);

	//Return imagen resultado
	return img_result;
}

extern __shared__ char sharedMem[];

__global__ void filter2D_Kernel_SM(char** in, char** out, float* mask, int width, int height)
{
	//Seleccionar un pixel de image_in
	int threadIDx = threadIdx.x + blockIdx.x * blockDim.x;
	int threadIDy = threadIdx.y + blockIdx.y * blockDim.y;

	if (threadIDx >= width) return;
	if (threadIDy >= height) return;

	int lindex = (threadIdx.y + 1) * (blockDim.x + 2) + (threadIdx.x + 1);

	((pixel32bpp*)sharedMem)[lindex] = ((pixel32bpp*)in[threadIDy])[threadIDx];

	if (threadIdx.x == 0 && threadIDx != 0) {
		((pixel32bpp*)sharedMem)[lindex - 1] = ((pixel32bpp*)in[threadIDy])[threadIDx - 1];
		if (threadIdx.y == blockDim.y - 1 && threadIDy != height - 1) {
			((pixel32bpp*)sharedMem)[lindex + (blockDim.x + 2) - 1] = ((pixel32bpp*)in[threadIDy + 1])[threadIDx - 1];
		}
	}

	if (threadIdx.x == blockDim.x - 1 && threadIDx != width - 1) {
		((pixel32bpp*)sharedMem)[lindex + 1] = ((pixel32bpp*)in[threadIDy])[threadIDx + 1];
		if (threadIdx.y == 0 && threadIDy != 0) {
			((pixel32bpp*)sharedMem)[lindex - (blockDim.x + 2) + 1] = ((pixel32bpp*)in[threadIDy - 1])[threadIDx + 1];
		}
	}

	if (threadIdx.y == 0 && threadIDy != 0) {
		((pixel32bpp*)sharedMem)[lindex - (blockDim.x + 2)] = ((pixel32bpp*)in[threadIDy - 1])[threadIDx];
		if (threadIdx.x == 0 && threadIDx != 0) {
			((pixel32bpp*)sharedMem)[lindex - (blockDim.x + 2) - 1] = ((pixel32bpp*)in[threadIDy - 1])[threadIDx - 1];
		}
	}

	if (threadIdx.y == blockDim.y - 1 && threadIDy != height - 1) {
		((pixel32bpp*)sharedMem)[lindex + (blockDim.x + 2)] = ((pixel32bpp*)in[threadIDy + 1])[threadIDx];
		if (threadIdx.x == blockDim.x - 1 && threadIDx != width - 1) {
			((pixel32bpp*)sharedMem)[lindex + (blockDim.x + 2) + 1] = ((pixel32bpp*)in[threadIDy + 1])[threadIDx + 1];
		}
	}
	
	if (threadIDx == 0 || threadIDx == width - 1)	return;
	if (threadIDy == 0 || threadIDy == height - 1)	return;

	__syncthreads();

	//Aplicar filtro a pixel
	pixel32bpp pixelIn, pixelOut;
	float a, r, g, b;
	a = r = g = b = 0;
	for (int i = -1; i <= 1; i++)
	{
		for (int j = -1; j <= 1; j++)
		{
			pixelIn = ((pixel32bpp*)sharedMem)[(threadIdx.y + 1 + i) * (blockDim.x + 2) + (threadIdx.x + 1 + j)];
			a += ((float)pixelIn.a)*mask[(i + 1) * 3 + j + 1];
			r += ((float)pixelIn.r)*mask[(i + 1) * 3 + j + 1];
			g += ((float)pixelIn.g)*mask[(i + 1) * 3 + j + 1];
			b += ((float)pixelIn.b)*mask[(i + 1) * 3 + j + 1];
		}
	}
	
	//Guardar resultado en image_out
	pixelOut.r = (char)r;
	pixelOut.g = (char)g;
	pixelOut.b = (char)b;
	pixelOut.a = (char)a;
	((pixel32bpp*)out[threadIDy])[threadIDx] = pixelOut;
}

Image_t* filter2D_CUDA_SM(Image_t* img, float* mask, int numThreadsKernel)
{
	//Reservar imagen resultado
	Image_t* img_result = (Image_t*)malloc(sizeof(Image_t));
	img_result->height = img->height;
	img_result->width = img->width;
	img_result->color_type = img->color_type;
	img_result->bit_depth = img->bit_depth;
	img_result->row_pointers = (char**)malloc(sizeof(char*)*img->height);
	for (int i = 0; i < img->height; i++)
	{
		img_result->row_pointers[i] = (char*)malloc(sizeof(char)*img->width * 4);
		memset(img_result->row_pointers[i], 0, sizeof(char)*img->width * 4);
	}

	//Copiar datos a GPU
	high_resolution_clock::time_point t1 = high_resolution_clock::now();
		//reservar memoria para la imagen resultado
	char** row_pointers_result = (char**)malloc(sizeof(char*)*img->height);
	for (int i = 0; i < img->height; i++)
	{
		hipMalloc((void**)&row_pointers_result[i], sizeof(char)*img->width * 4);
		hipMemset(row_pointers_result[i], 0, sizeof(char)*img->width * 4);
		cudaCheckError();
	}
	char** d_pixels_result = NULL;
	hipMalloc((void**)&d_pixels_result, sizeof(char*)*img->height);
	hipMemset(d_pixels_result, 0, sizeof(char*)*img->height);
	hipMemcpy(d_pixels_result, row_pointers_result, sizeof(char*)*img->height, hipMemcpyHostToDevice);
	cudaCheckError();

	//reservar memoria y copiar imagen original
	char** row_pointers = (char**)malloc(sizeof(char*)*img->height);
	for (int i = 0; i < img->height; i++)
	{
		hipMalloc((void**)&row_pointers[i], sizeof(char)*img->width * 4);
		hipMemset(row_pointers[i], 0, sizeof(char)*img->width * 4);
		hipMemcpy(row_pointers[i], img->row_pointers[i], sizeof(char)*img->width * 4, hipMemcpyHostToDevice);
		cudaCheckError();
	}
	char** d_pixels = NULL;
	hipMalloc((void**)&d_pixels, sizeof(char*)*img->height);
	hipMemset(d_pixels, 0, sizeof(char*)*img->height);
	hipMemcpy(d_pixels, row_pointers, sizeof(char*)*img->height, hipMemcpyHostToDevice);
	cudaCheckError();

	//copiar el filtro a GPU
	float* d_mask;
	hipMalloc((void**)&d_mask, 9 * sizeof(float));
	hipMemset(d_mask, 0, sizeof(float) * 9);
	hipMemcpy(d_mask, mask, sizeof(float) * 9, hipMemcpyHostToDevice);
	cudaCheckError();
	duration<double> time_span = duration_cast<duration<double>>(high_resolution_clock::now() - t1);
	std::cout << "\t Tiempo de copia de datos de CPU a GPU: " << time_span.count() * 1000 << "\n";

	//Ejecutar filter2D_Kernel_1
	dim3 threadsPerBlock = dim3(numThreadsKernel, numThreadsKernel, 1);
	dim3 numBlocks = dim3((img->width / threadsPerBlock.x) + 1, (img->height / threadsPerBlock.y) + 1, 1);

	t1 = high_resolution_clock::now();
	filter2D_Kernel_SM << <numBlocks, threadsPerBlock, (numThreadsKernel + 2)*(numThreadsKernel + 2) * 4>> >
		(d_pixels, d_pixels_result, d_mask, img->width, img->height);
	cudaCheckError();
	hipDeviceSynchronize();
	cudaCheckError();
	time_span = duration_cast<duration<double>>(high_resolution_clock::now() - t1);
	std::cout << "\t Tiempo de ejecucion del kernel: " << time_span.count() * 1000 << "\n";

	//Copiar datos de GPU
	t1 = high_resolution_clock::now();
	for (int i = 0; i < img->height; i++)
	{
		hipMemcpy(img_result->row_pointers[i], row_pointers_result[i], sizeof(char) * img->width * 4, hipMemcpyDeviceToHost);
	}
	cudaCheckError();
	time_span = duration_cast<duration<double>>(high_resolution_clock::now() - t1);
	std::cout << "\t Tiempo de copia de datos de GPU a CPU: " << time_span.count() * 1000 << "\n";

	//Liberar memoria
	for (int i = 0; i < img->height; i++)
	{
		hipFree(row_pointers_result[i]);
		hipFree(row_pointers[i]);
	}
	free(row_pointers_result);
	free(row_pointers);
	hipFree(d_pixels_result);
	hipFree(d_pixels);
	hipFree(d_mask);

	//Return imagen resultado
	return img_result;
}

Image_t* filter2D_CUDA_Async(Image_t* img, float* mask, int numThreadsKernel)
{
	//Crear stream
	hipStream_t stream1;
	hipStreamCreate(&stream1);

	//Reservar imagen resultado
	Image_t* img_result = (Image_t*)malloc(sizeof(Image_t));
	img_result->height = img->height;
	img_result->width = img->width;
	img_result->color_type = img->color_type;
	img_result->bit_depth = img->bit_depth;
	img_result->row_pointers = (char**)malloc(sizeof(char*)*img->height);
	for (int i = 0; i < img->height; i++)
	{
		img_result->row_pointers[i] = (char*)malloc(sizeof(char)*img->width * 4);
		memset(img_result->row_pointers[i], 0, sizeof(char)*img->width * 4);
	}

	//Copiar datos a GPU
	high_resolution_clock::time_point t1 = high_resolution_clock::now();
		//reservar memoria para la imagen resultado
	char** row_pointers_result = (char**)malloc(sizeof(char*)*img->height);
	for (int i = 0; i < img->height; i++)
	{
		hipMalloc((void**)&row_pointers_result[i], sizeof(char)*img->width * 4);
		hipMemsetAsync(row_pointers_result[i], 0, sizeof(char)*img->width * 4, stream1);
	}
	char** d_pixels_result = NULL;
	hipMalloc((void**)&d_pixels_result, sizeof(char*)*img->height);
	hipMemsetAsync(d_pixels_result, 0, sizeof(char*)*img->height, stream1);
	hipMemcpyAsync(d_pixels_result, row_pointers_result, sizeof(char*)*img->height, hipMemcpyHostToDevice, stream1);

	//reservar memoria y copiar imagen original
	char** row_pointers = (char**)malloc(sizeof(char*)*img->height);
	for (int i = 0; i < img->height; i++)
	{
		hipMalloc((void**)&row_pointers[i], sizeof(char)*img->width * 4);
		hipMemsetAsync(row_pointers[i], 0, sizeof(char)*img->width * 4, stream1);
		hipMemcpyAsync(row_pointers[i], img->row_pointers[i], sizeof(char)*img->width * 4, hipMemcpyHostToDevice, stream1);
	}
	char** d_pixels = NULL;
	hipMalloc((void**)&d_pixels, sizeof(char*)*img->height);
	hipMemsetAsync(d_pixels, 0, sizeof(char*)*img->height, stream1);
	hipMemcpyAsync(d_pixels, row_pointers, sizeof(char*)*img->height, hipMemcpyHostToDevice, stream1);

	//copiar el filtro a GPU
	float* d_mask;
	hipMalloc((void**)&d_mask, 9 * sizeof(float));
	hipMemsetAsync(d_mask, 0, sizeof(float) * 9, stream1);
	hipMemcpyAsync(d_mask, mask, sizeof(float) * 9, hipMemcpyHostToDevice, stream1);
	duration<double> time_span = duration_cast<duration<double>>(high_resolution_clock::now() - t1);
	std::cout << "\t Tiempo de copia de datos de CPU a GPU: " << time_span.count() * 1000 << "\n";

	//Ejecutar filter2D_Kernel_SM
	dim3 threadsPerBlock = dim3(numThreadsKernel, numThreadsKernel, 1);
	dim3 numBlocks = dim3((img->width / threadsPerBlock.x) + 1, (img->height / threadsPerBlock.y) + 1, 1);

	t1 = high_resolution_clock::now();
	filter2D_Kernel_1 << <numBlocks, threadsPerBlock, 0, stream1 >> >
		(d_pixels, d_pixels_result, d_mask, img->width, img->height);
	time_span = duration_cast<duration<double>>(high_resolution_clock::now() - t1);
	std::cout << "\t Tiempo de ejecucion del kernel: " << time_span.count() * 1000 << "\n";

	//Copiar datos de GPU
	t1 = high_resolution_clock::now();
	for (int i = 0; i < img->height; i++)
	{
		hipMemcpyAsync(img_result->row_pointers[i], row_pointers_result[i], sizeof(char) * img->width * 4, hipMemcpyDeviceToHost, stream1);
	}
	time_span = duration_cast<duration<double>>(high_resolution_clock::now() - t1);
	std::cout << "\t Tiempo de copia de datos de GPU a CPU: " << time_span.count() * 1000 << "\n";

	//Sincronizar memoria
	hipStreamSynchronize(stream1);
	cudaCheckError();

	//Liberar memoria
	for (int i = 0; i < img->height; i++)
	{
		hipFree(row_pointers_result[i]);
		hipFree(row_pointers[i]);
	}
	free(row_pointers_result);
	free(row_pointers);
	hipFree(d_pixels_result);
	hipFree(d_pixels);
	hipFree(d_mask);

	//Return imagen resultado
	return img_result;
}

Image_t* filter2D(Image_t* im, float* mask)
{
	Image_t* imOut = new Image_t;
	(*imOut) = (*im);
	imOut->row_pointers = (char**)malloc(sizeof(char*)*im->height);
	for (int i = 0; i < im->height; i++)
	{
		imOut->row_pointers[i] = (char*)malloc(sizeof(char)*im->width * 4);
		memset(imOut->row_pointers[i], 0, sizeof(char)*im->width * 4);
	}

	for (int i = 1; i < (im->height - 1); i++)
		for (int j = 1; j < (im->width - 1); j++)
		{
			pixel32bpp pixlIn;
			float a, r, g, b;
			a = r = g = b = 0;
			for (int y = -1; y < 2; y++)
				for (int x = -1; x < 2; x++)
				{
					pixlIn = *((pixel32bpp*)(&(im->row_pointers[i + y][(j + x) * 4])));
					a += ((float)pixlIn.a)*mask[(y + 1) * 3 + x + 1];
					r += ((float)pixlIn.r)*mask[(y + 1) * 3 + x + 1];
					g += ((float)pixlIn.g)*mask[(y + 1) * 3 + x + 1];
					b += ((float)pixlIn.b)*mask[(y + 1) * 3 + x + 1];
				}
			imOut->row_pointers[i][j * 4] = (char)r;
			imOut->row_pointers[i][j * 4 + 1] = (char)g;
			imOut->row_pointers[i][j * 4 + 2] = (char)b;
			imOut->row_pointers[i][j * 4 + 3] = (char)a;
		}
	return imOut;
}